#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <random>
#include "Timer.cuh"

using namespace timer;
using namespace timer_cuda;

const int BLOCK_SIZE = 512;
__device__ int block_counter;

__global__ void PrefixScan(int* VectorIN, int N) {
	int step, limit;
	int valueRight, valueLeft;

	step = 1;
	for (limit = blockDim.x / 2; limit > 0; limit /= 2) {
		if (threadIdx.x < limit) {
			valueRight = (threadIdx.x + 1) * (step * 2) - 1;
			valueLeft = valueRight - step;
			VectorIN[valueRight] = VectorIN[valueRight] + VectorIN[valueLeft];
		}
		step *= 2;
		__syncthreads();
	}
	
	if (threadIdx.x == 0)
		VectorIN[blockDim.x - 1] = 0;
	__syncthreads();

	limit = 1;	
	for (step = blockDim.x / 2; step > 0; step /= 2) {
		if (threadIdx.x < limit) {
			valueRight = (threadIdx.x * 2 + 1) * step - 1;
			valueLeft = valueRight - step;
			int tmp = VectorIN[valueLeft];
			VectorIN[valueLeft] = VectorIN[valueRight];
			VectorIN[valueRight] = tmp + VectorIN[valueRight];
		}
		limit *= 2;
		__syncthreads();
	}
}

void printArray(int* Array, int N, const char str[] = "") {
	std::cout << str;
	for (int i = 0; i < N; ++i)
		std::cout << std::setw(5) << Array[i] << ' ';
	std::cout << std::endl << std::endl;
}

void printArray(int* Array, int start, int end, const char str[] = "") {
	std::cout << str;
	for (int i = start; i < end; ++i)
		std::cout << std::setw(5) << Array[i] << ' ';
	std::cout << std::endl << std::endl;
}

#define DIV(a,b)	(((a) + (b) - 1) / (b))

int main() {
	const int blockDim = BLOCK_SIZE;
	const int N = BLOCK_SIZE * 131072;
	
    // ------------------- INIT ------------------------------------------------

    // Random Engine Initialization
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator (seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    timer::Timer<HOST> host_TM;
    timer_cuda::Timer<DEVICE> dev_TM;

	// ------------------ HOST INIT --------------------------------------------

	int* VectorIN = new int[N];
	for (int i = 0; i < N; ++i)
		VectorIN[i] = distribution(generator);

	// ------------------- CUDA INIT -------------------------------------------

	int* devVectorIN;
	__SAFE_CALL( hipMalloc(&devVectorIN, N * sizeof(int)) );
  __SAFE_CALL( hipMemcpy(devVectorIN, VectorIN, N * sizeof(int), hipMemcpyHostToDevice) );

	int* prefixScan = new int[N];
	float dev_time;

	// ------------------- CUDA COMPUTATION 1 ----------------------------------

	dev_TM.start();
	PrefixScan<<<DIV(N, blockDim), blockDim>>>(devVectorIN, N);
	dev_TM.stop();
	dev_time = dev_TM.duration();

	printArray(devVectorIN, 10);
	__SAFE_CALL(hipMemcpy(prefixScan, devVectorIN, N * sizeof(int), hipMemcpyDeviceToHost) );

	// ------------------- CUDA ENDING -----------------------------------------

	std::cout << std::fixed << std::setprecision(1) << "KernelTime Naive  : " << dev_time << std::endl << std::endl;

	// ------------------- VERIFY ----------------------------------------------

    host_TM.start();

	int* host_result = new int[N];
	std::partial_sum(VectorIN, VectorIN + N, host_result);

    host_TM.stop();

	if (!std::equal(host_result, host_result + blockDim - 1, prefixScan + 1)) {
		std::cerr << " Error! :  prefixScan" << std::endl << std::endl;
		hipDeviceReset();
		std::exit(EXIT_FAILURE);
	}

    // ----------------------- SPEEDUP -----------------------------------------

    float speedup1 = host_TM.duration() / dev_time;
	std::cout << "Correct result" << std::endl
              << "(1) Speedup achieved: " << speedup1 << " x" << std::endl
              << std::endl << std::endl;

    std::cout << host_TM.duration() << ";" << dev_TM.duration() << ";" << host_TM.duration() / dev_TM.duration() << std::endl;
	
	delete[] host_result;
    delete[] prefixScan;
    
    __SAFE_CALL( hipFree(devVectorIN) );
    
    hipDeviceReset();
}
